#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "glog/logging.h"
#include <cub/block/block_reduce.cuh>

#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t status = call;                                                                                     \
        if (status != hipSuccess) {                                                                                   \
            LOG(FATAL) << "CUDA Error: " << hipGetErrorString(status) << " at " << __FILE__ << ":" << __LINE__;       \
        }                                                                                                              \
    } while (0)

#define CUBLAS_CHECK(call)                                                                                             \
    do {                                                                                                               \
        hipblasStatus_t status = call;                                                                                  \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                                                         \
            LOG(FATAL) << "CUBLAS Error: " << cublasGetStatusString(status) << " at " << __FILE__ << ":" << __LINE__;  \
        }                                                                                                              \
    } while (0)

std::shared_ptr<Tensor> MatmulForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other) {
    // =================================== 作业 ===================================
    // TODO：实现CUDA上的矩阵乘法前向计算
    // REF:
    // =================================== 作业 ===================================
    // 参考 CPU 实现，支持批量维度：假设 input[..., M, K] 与 other[..., K, N]
    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();
    CHECK_GE(input_dims.size(), 2);
    CHECK_GE(other_dims.size(), 2);
    CHECK_EQ(input_dims.back(), other_dims[other_dims.size() - 2]); // K 对齐

    // 计算 batch 数：逐 batch 做 [M,K] x [K,N]
    const int64_t M = input_dims[input_dims.size() - 2];
    const int64_t K = input_dims.back();
    const int64_t N = other_dims.back();

    const int64_t num_batches = input->NumElements() / (M * K);
    CHECK_EQ(other->NumElements(), num_batches * K * N); // 保证 batch 对齐

    auto output_dims = input_dims;
    output_dims.back() = N;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());

    const float alpha = 1.0f;
    const float beta = 0.0f;
    // 利用 row-major -> column-major 转换：
    // C_row (M,N) -> C_col (N,M) = B_row^T (N,K) * A_row^T (K,M)
    // 于是调用: m=N, n=M, k=K, A=other(视作 N x K), B=input(视作 K x M)
    hipblasHandle_t handle; CUBLAS_CHECK(hipblasCreate(&handle));

    const int64_t strideA = K * N; // other 每个 batch 大小 (K,N) row-major == (N,K) col-major
    const int64_t strideB = M * K; // input 每个 batch
    const int64_t strideC = M * N; // output 每个 batch (row) == (N,M) col

    CUBLAS_CHECK(hipblasSgemmStridedBatched(handle,
                                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                                           (int)N, (int)M, (int)K,
                                           &alpha,
                                           static_cast<const float*>(other->DataPtr()), (int)N, strideA,
                                           static_cast<const float*>(input->DataPtr()), (int)K, strideB,
                                           &beta,
                                           static_cast<float*>(output->DataPtr()), (int)N, strideC,
                                           (int)num_batches));

    CUBLAS_CHECK(hipblasDestroy(handle));
    return output;
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
MatmulBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &other,
               const std::shared_ptr<Tensor> &grad_output) {
    // =================================== 作业 ===================================
    // TODO：实现CUDA上的矩阵乘法反向传播
    // REF:
    // =================================== 作业 ===================================
    // grad_input = grad_output * other^T
    // grad_other = input^T * grad_output
    const auto &input_dims = input->Dims();
    const auto &other_dims = other->Dims();
    const auto &grad_dims = grad_output->Dims();
    CHECK_GE(input_dims.size(), 2);
    CHECK_GE(other_dims.size(), 2);
    CHECK_GE(grad_dims.size(), 2);

    const int64_t M = input_dims[input_dims.size() - 2];
    const int64_t K = input_dims.back();
    const int64_t N = other_dims.back();
    CHECK_EQ(grad_dims[grad_dims.size() - 2], M);
    CHECK_EQ(grad_dims.back(), N);
    CHECK_EQ(other_dims[other_dims.size() - 2], K);

    const int64_t num_batches = input->NumElements() / (M * K);
    CHECK_EQ(other->NumElements(), num_batches * K * N);
    CHECK_EQ(grad_output->NumElements(), num_batches * M * N);

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, input->GetDevice());
    auto grad_other = std::make_shared<Tensor>(other_dims, DataType::kFLOAT32, other->GetDevice());

    const float alpha = 1.0f;
    const float beta0 = 0.0f;
    hipblasHandle_t handle; CUBLAS_CHECK(hipblasCreate(&handle));

    // 1) grad_input_row (M,K): compute grad_input_row^T (K,M) = other_row * grad_output_row^T
    // 使用: C = op(A)*op(B)
    // A: other buffer 视作 (N,K) col-major, op(A)=T -> (K,N)
    // B: grad_output buffer 视作 (N,M) col-major, op(B)=N -> (N,M)
    // C: grad_input^T (K,M)
    const int64_t strideOther = K * N;
    const int64_t strideGradOut = M * N;
    const int64_t strideGradIn = M * K;
    CUBLAS_CHECK(hipblasSgemmStridedBatched(handle,
                                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                                           (int)K, (int)M, (int)N,
                                           &alpha,
                                           static_cast<const float*>(other->DataPtr()), (int)N, strideOther,
                                           static_cast<const float*>(grad_output->DataPtr()), (int)N, strideGradOut,
                                           &beta0,
                                           static_cast<float*>(grad_input->DataPtr()), (int)K, strideGradIn,
                                           (int)num_batches));

    // 2) grad_other_row (K,N): compute grad_other_row^T (N,K) = grad_output_row^T (N,M) * input_row (M,K)
    const int64_t strideInput = M * K;
    const int64_t strideGradOther = K * N;
    CUBLAS_CHECK(hipblasSgemmStridedBatched(handle,
                                           HIPBLAS_OP_N, HIPBLAS_OP_T,
                                           (int)N, (int)K, (int)M,
                                           &alpha,
                                           static_cast<const float*>(grad_output->DataPtr()), (int)N, strideGradOut,
                                           static_cast<const float*>(input->DataPtr()), (int)K, strideInput,
                                           &beta0,
                                           static_cast<float*>(grad_other->DataPtr()), (int)N, strideGradOther,
                                           (int)num_batches));

    CUBLAS_CHECK(hipblasDestroy(handle));
    return {grad_input, grad_other};
}

__global__ void BiasCopyKernel(float *output, const float *bias, int bs, int out_features) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= bs * out_features) {
        return;
    }
    int j = idx % out_features;
    output[idx] = bias[j];
}

std::shared_ptr<Tensor> LinearForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight,
                                      bool transpose, const std::shared_ptr<Tensor> &bias) {

    /*
        !transpose: output = input * weight + bias
        output[*, out_features] = input[*, in_features] * weight[in_features, out_features] + bias[out_features]

        transpose:  output = input * weight^T + bias
        output[*, out_features] = input[*, in_features] * weight[out_features, in_features]^T + bias[out_features]
    */

    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);

    // As for cublas:
    // C = alpha * op(B) * op(A) + beta * C
    // Dimensions:
    //   input:  (bs, in_features)
    //   weight: (in_features, out_features) or (out_features, in_features) if transposed
    //   output: (bs, out_features)
    const int64_t out_features = weight_dims[transpose ? 0 : 1];

    auto output_dims = input_dims;
    *output_dims.rbegin() = out_features;
    auto output = std::make_shared<Tensor>(output_dims, DataType::kFLOAT32, input->GetDevice());

    if (bias) {
        CHECK_EQ(bias->Dims().size(), 1);
        CHECK_EQ(bias->Dims()[0], out_features);
        int threads_per_block = 256;
        int num_blocks = (bs * out_features + threads_per_block - 1) / threads_per_block;
        BiasCopyKernel<<<num_blocks, threads_per_block>>>(
            static_cast<float *>(output->DataPtr()), static_cast<const float *>(bias->DataPtr()), bs, out_features);
    } else {
        output->Fill<float>(0.0f);
    }

    const float alpha = 1.0f;
    const float beta = 1.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    if (transpose) {
        // weight is [out_features, in_features] here

        // output = input * weight.T --> output.T = weight * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[in_features, out_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    } else {
        // output = input * weight --> output.T =  weight.T * input.T
        // C = output.T[out_features, bs]
        // A = weight.T[out_features, in_features]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, out_features, bs, in_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(output->DataPtr()), out_features));
    }
    CUBLAS_CHECK(hipblasDestroy(handle));
    return output;
}

template <int BLOCK_SIZE>
__global__ void ReduceColumnsKernel(const float *__restrict__ input, float *__restrict__ output, int num_rows,
                                    int num_cols) {
    using BlockReduce = hipcub::BlockReduce<float, BLOCK_SIZE>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int row = blockIdx.x;
    float sum = 0.0f;

    for (int col = threadIdx.x; col < num_cols; col += blockDim.x) { sum += input[row * num_cols + col]; }

    float reduced = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0) {
        output[row] = reduced;
    }
}

std::tuple<std::shared_ptr<Tensor>, std::shared_ptr<Tensor>, std::shared_ptr<Tensor>>
LinearBackward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &weight, bool transpose,
               int64_t out_features, const std::shared_ptr<Tensor> &grad_output, const bool bias) {
    const auto &input_dims = input->Dims();
    CHECK_GE(input_dims.size(), 2);
    const int64_t bs = std::accumulate(input_dims.rbegin() + 1, input_dims.rend(), 1, std::multiplies<int64_t>{});
    const int64_t in_features = *input_dims.rbegin();

    const auto &weight_dims = weight->Dims();
    CHECK_EQ(weight_dims.size(), 2);
    CHECK_EQ(in_features, weight_dims[transpose ? 1 : 0]);
    CHECK_EQ(out_features, weight_dims[transpose ? 0 : 1]);

    auto grad_input = std::make_shared<Tensor>(input_dims, DataType::kFLOAT32, grad_output->GetDevice());
    auto grad_weight = std::make_shared<Tensor>(weight_dims, DataType::kFLOAT32, grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);
    grad_weight->Fill<float>(0.0f);
    std::shared_ptr<Tensor> grad_bias = nullptr;
    if (bias) {
        grad_bias = std::make_shared<Tensor>(std::vector<int64_t>{out_features}, DataType::kFLOAT32,
                                             grad_output->GetDevice());
        grad_bias->Fill<float>(0.0f);
    }

    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    if (transpose) {
        // weight is [out_features, in_features] here

        // d_input = d_output * weight --> d_input.T = weight.T * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[in_features, out_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = d_output.T * input --> d_weight.T = input.T * d_output
        // C = d_weight.T[in_features, out_features]
        // A = input.T[in_features, bs]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, in_features, out_features, bs, &alpha,
                                 static_cast<const float *>(input->DataPtr()), in_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), in_features));
    } else {
        // weight is [in_features, out_features] here

        // d_input = d_output * weight.T --> d_input.T = weight * d_output.T
        // C = d_input.T[in_features, bs]
        // A = weight.T[out_features, in_features]
        // B = d_output.T[out_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, bs, out_features, &alpha,
                                 static_cast<const float *>(weight->DataPtr()), out_features,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features, &beta,
                                 static_cast<float *>(grad_input->DataPtr()), in_features));

        // d_weight = input.T * d_output --> d_weight.T = d_output.T * input
        // C = d_weight.T[out_features, in_features]
        // A = d_output.T[out_features, bs]
        // B = input.T[in_features, bs]
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, out_features, in_features, bs, &alpha,
                                 static_cast<const float *>(grad_output->DataPtr()), out_features,
                                 static_cast<const float *>(input->DataPtr()), in_features, &beta,
                                 static_cast<float *>(grad_weight->DataPtr()), out_features));
    }

    // d_bias = \sum_i(i=0, bs-1) d_output[i]
    if (bias) {
        constexpr int BLOCK_SIZE = 256;
        int threads_per_block = BLOCK_SIZE;
        int num_blocks = out_features;
        ReduceColumnsKernel<BLOCK_SIZE>
            <<<num_blocks, threads_per_block>>>(static_cast<const float *>(grad_output->DataPtr()),
                                                static_cast<float *>(grad_bias->DataPtr()), out_features, bs);
    }

    CUBLAS_CHECK(hipblasDestroy(handle));

    return {grad_input, grad_weight, grad_bias};
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_LINEAR_KERNEL(kernel_name)                                                                       \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_LINEAR_KERNEL(MatmulForward)
REGISTER_CUDA_LINEAR_KERNEL(MatmulBackward)
REGISTER_CUDA_LINEAR_KERNEL(LinearForward)
REGISTER_CUDA_LINEAR_KERNEL(LinearBackward)

#undef REGISTER_CUDA_LINEAR_KERNEL
