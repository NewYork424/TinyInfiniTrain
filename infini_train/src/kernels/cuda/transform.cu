#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "glog/logging.h"

#include "infini_train/include/dispatcher.h"
#include "infini_train/include/tensor.h"

namespace infini_train::kernels::cuda {

__global__ void TrilForwardKernel(const float *input, float *output, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal >= 0) {
        output[idx] = input[idx];
    } else {
        output[idx] = 0.0f;
    }
}

std::shared_ptr<Tensor> TrilForward(const std::shared_ptr<Tensor> &input, int64_t diagonal) {
    CHECK_EQ(input->Dims().size(), 2);
    int64_t rows = input->Dims()[0];
    int64_t cols = input->Dims()[1];

    auto output = std::make_shared<Tensor>(input->Dims(), input->Dtype(), input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    TrilForwardKernel<<<num_blocks, threads_per_block>>>(static_cast<float *>(input->DataPtr()),
                                                         static_cast<float *>(output->DataPtr()), rows, cols, diagonal);
    return output;
}

__global__ void TrilBackwardKernel(const float *grad_output, float *grad_input, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal >= 0) {
        grad_input[idx] = grad_output[idx];
    } else {
        grad_input[idx] = 0.0f;
    }
}

std::shared_ptr<Tensor> TrilBackward(const std::shared_ptr<Tensor> &grad_output, int64_t diagonal) {
    int rows = grad_output->Dims()[0];
    int cols = grad_output->Dims()[1];

    auto grad_input = std::make_shared<Tensor>(grad_output->Dims(), grad_output->Dtype(), grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    TrilBackwardKernel<<<num_blocks, threads_per_block>>>(static_cast<const float *>(grad_output->DataPtr()),
                                                          static_cast<float *>(grad_input->DataPtr()), rows, cols,
                                                          diagonal);

    return grad_input;
}

__global__ void TriuForwardKernel(const float *input, float *output, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal <= 0) {
        output[idx] = input[idx];
    } else {
        output[idx] = 0.0f;
    }
}

std::shared_ptr<Tensor> TriuForward(const std::shared_ptr<Tensor> &input, int64_t diagonal) {
    CHECK_EQ(input->Dims().size(), 2);
    int64_t rows = input->Dims()[0];
    int64_t cols = input->Dims()[1];

    auto output = std::make_shared<Tensor>(input->Dims(), input->Dtype(), input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    TriuForwardKernel<<<num_blocks, threads_per_block>>>(static_cast<const float *>(input->DataPtr()),
                                                         static_cast<float *>(output->DataPtr()), rows, cols, diagonal);

    return output;
}

__global__ void TriuBackwardKernel(const float *grad_output, float *grad_input, int rows, int cols, int64_t diagonal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= rows * cols) {
        return;
    }

    int row = idx / cols;
    int col = idx % cols;

    if (row - col + diagonal <= 0) {
        grad_input[idx] = grad_output[idx];
    } else {
        grad_input[idx] = 0.0f;
    }
}

std::shared_ptr<Tensor> TriuBackward(const std::shared_ptr<Tensor> &grad_output, int64_t diagonal) {
    int rows = grad_output->Dims()[0];
    int cols = grad_output->Dims()[1];

    auto grad_input = std::make_shared<Tensor>(grad_output->Dims(), grad_output->Dtype(), grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);

    int threads_per_block = 256;
    int num_blocks = (rows * cols + threads_per_block - 1) / threads_per_block;

    TriuBackwardKernel<<<num_blocks, threads_per_block>>>(static_cast<const float *>(grad_output->DataPtr()),
                                                          static_cast<float *>(grad_input->DataPtr()), rows, cols,
                                                          diagonal);

    return grad_input;
}

__global__ void TransposeForwardKernel(const float *input, float *output, const int64_t *in_dims,
                                       const int64_t *in_strides, const int64_t *out_strides, int64_t ndim,
                                       int64_t dim0, int64_t dim1, int64_t num_elements) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_elements) {
        return;
    }

    int64_t remaining = idx;
    int64_t coords[8];

    // 1. decode coord from output index
    for (int i = 0; i < ndim; ++i) {
        coords[i] = remaining / out_strides[i];
        remaining %= out_strides[i];
    }

    // 2. swap the coordinates
    int64_t tmp = coords[dim0];
    coords[dim0] = coords[dim1];
    coords[dim1] = tmp;

    // 3. compute input flat index
    int64_t in_flat_idx = 0;
    for (int i = 0; i < ndim; ++i) { in_flat_idx += coords[i] * in_strides[i]; }

    output[idx] = input[in_flat_idx];
}

std::shared_ptr<Tensor> TransposeForward(const std::shared_ptr<Tensor> &input, int64_t dim0, int64_t dim1) {
    CHECK_LE(input->Dims().size(), 8);
    dim0 = dim0 < 0 ? dim0 + input->Dims().size() : dim0;
    dim1 = dim1 < 0 ? dim1 + input->Dims().size() : dim1;
    CHECK(dim0 >= 0 && dim0 < input->Dims().size() && dim1 >= 0 && dim1 < input->Dims().size());

    auto in_dims = input->Dims();
    std::vector<int64_t> out_dims = in_dims;
    std::swap(out_dims[dim0], out_dims[dim1]);

    auto output = std::make_shared<Tensor>(out_dims, input->Dtype(), input->GetDevice());
    output->Fill<float>(0.0f);
    int64_t ndim = in_dims.size();
    int64_t num_elements = output->NumElements();

    // compute strides of in_dims and out_dims
    std::vector<int64_t> in_strides(ndim, 1);
    std::vector<int64_t> out_strides(ndim, 1);
    for (int i = ndim - 2; i >= 0; --i) {
        in_strides[i] = in_strides[i + 1] * in_dims[i + 1];
        out_strides[i] = out_strides[i + 1] * out_dims[i + 1];
    }

    // Allocate device memory for dims and strides
    int64_t *device_buffer;
    hipMallocAsync(&device_buffer, 3 * ndim * sizeof(int64_t), 0);

    int64_t *in_dims_dev = device_buffer;
    int64_t *in_strides_dev = device_buffer + ndim;
    int64_t *out_strides_dev = device_buffer + 2 * ndim;

    std::vector<int64_t> host_buffer;
    host_buffer.insert(host_buffer.end(), in_dims.begin(), in_dims.end());
    host_buffer.insert(host_buffer.end(), in_strides.begin(), in_strides.end());
    host_buffer.insert(host_buffer.end(), out_strides.begin(), out_strides.end());

    hipMemcpyAsync(device_buffer, host_buffer.data(), 3 * ndim * sizeof(int64_t), hipMemcpyHostToDevice, 0);

    int threads_per_block = 256;
    int num_blocks = (num_elements + threads_per_block - 1) / threads_per_block;

    TransposeForwardKernel<<<num_blocks, threads_per_block>>>(
        static_cast<const float *>(input->DataPtr()), static_cast<float *>(output->DataPtr()), in_dims_dev,
        in_strides_dev, out_strides_dev, ndim, dim0, dim1, num_elements);

    hipFreeAsync(device_buffer, 0);

    return output;
}

std::shared_ptr<Tensor> TransposeBackward(const std::shared_ptr<Tensor> &grad_output, int64_t dim0, int64_t dim1) {
    return TransposeForward(grad_output, dim1, dim0);
}

__global__ void MaskForwardKernel(const float *input, const float *mask, float *output, float value, int batch_size,
                                  int mask_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < batch_size * mask_size) {
        output[i] = (mask[i % mask_size] == 1.0f) ? value : input[i];
    }
}

std::shared_ptr<Tensor> MaskForward(const std::shared_ptr<Tensor> &input, const std::shared_ptr<Tensor> &mask,
                                    float value) {
    auto input_shape = input->Dims();
    auto mask_shape = mask->Dims();
    CHECK_EQ(static_cast<int>(input->Dtype()), static_cast<int>(mask->Dtype()));

    int64_t input_dims = input_shape.size();
    int64_t mask_dims = mask_shape.size();
    for (int i = 0; i < mask_dims; ++i) {
        int input_dim = input_shape[input_dims - mask_dims + i];
        int mask_dim = mask_shape[i];
        CHECK(input_dim == mask_dim || mask_dim == 1);
    }

    int64_t mask_size = mask->NumElements();
    int64_t batch_size = input->NumElements() / mask_size;

    auto output = std::make_shared<Tensor>(input->Dims(), input->Dtype(), input->GetDevice());

    int threads_per_block = 256;
    int num_blocks = (input->NumElements() + threads_per_block - 1) / threads_per_block;

    MaskForwardKernel<<<num_blocks, threads_per_block>>>(
        static_cast<const float *>(input->DataPtr()), static_cast<const float *>(mask->DataPtr()),
        static_cast<float *>(output->DataPtr()), value, batch_size, mask_size);
    return output;
}

__global__ void MaskBackwardKernel(const float *grad_output, const float *mask, float *grad_input, int batch_size,
                                   int mask_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < batch_size * mask_size) {
        grad_input[i] = (mask[i % mask_size] == 1.0f) ? 0.0f : grad_output[i];
    }
}

std::shared_ptr<Tensor> MaskBackward(const std::shared_ptr<Tensor> &grad_output, const std::shared_ptr<Tensor> &mask) {
    auto output_shape = grad_output->Dims();
    auto mask_shape = mask->Dims();
    CHECK_EQ(static_cast<int>(grad_output->Dtype()), static_cast<int>(mask->Dtype()));

    int64_t output_dims = output_shape.size();
    int64_t mask_dims = mask_shape.size();
    for (int i = 0; i < mask_dims; ++i) {
        int out_dim = output_shape[output_dims - mask_dims + i];
        int mask_dim = mask_shape[i];
        CHECK(out_dim == mask_dim || mask_dim == 1);
    }

    int64_t mask_size = mask->NumElements();
    int64_t batch_size = grad_output->NumElements() / mask_size;

    auto grad_input = std::make_shared<Tensor>(grad_output->Dims(), grad_output->Dtype(), grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);

    int threads_per_block = 256;
    int num_blocks = (grad_output->NumElements() + threads_per_block - 1) / threads_per_block;

    MaskBackwardKernel<<<num_blocks, threads_per_block>>>(
        static_cast<const float *>(grad_output->DataPtr()), static_cast<const float *>(mask->DataPtr()),
        static_cast<float *>(grad_input->DataPtr()), batch_size, mask_size);
    return grad_input;
}

__global__ void RepeatInterleaveForwardKernel(const float *input, float *output, int64_t outer, int64_t dim_size,
                                              int64_t inner, int64_t repeat) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total = outer * dim_size * repeat * inner;
    if (idx >= total) {
        return;
    }

    int64_t i = idx / inner;
    int64_t j = idx % inner;

    int64_t o = i / (dim_size * repeat);
    int64_t di = (i / repeat) % dim_size;

    output[idx] = input[(o * dim_size + di) * inner + j];
}

std::shared_ptr<Tensor> RepeatInterleaveForward(const std::shared_ptr<Tensor> &input, int64_t repeat, int64_t dim) {
    CHECK_GT(repeat, 0);
    CHECK_GE(dim, 0);
    CHECK_LT(dim, input->Dims().size());

    const auto &input_dims = input->Dims();
    const int64_t outer = std::accumulate(input_dims.begin(), input_dims.begin() + dim, 1, std::multiplies<int64_t>());
    const int64_t inner
        = std::accumulate(input_dims.begin() + dim + 1, input_dims.end(), 1, std::multiplies<int64_t>());
    const int64_t dim_size = input_dims[dim];

    std::vector<int64_t> output_dims = input_dims;
    output_dims[dim] = dim_size * repeat;
    auto output = std::make_shared<Tensor>(output_dims, input->Dtype(), input->GetDevice());

    const float *input_ptr = static_cast<const float *>(input->DataPtr());
    float *output_ptr = static_cast<float *>(output->DataPtr());

    int64_t total_elements = outer * dim_size * repeat * inner;
    int threads_per_block = 256;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;
    RepeatInterleaveForwardKernel<<<num_blocks, threads_per_block>>>(input_ptr, output_ptr, outer, dim_size, inner,
                                                                     repeat);

    return output;
}

__global__ void RepeatInterleaveBackwardKernel(const float *grad_output, float *grad_input, int64_t outer,
                                               int64_t dim_size, int64_t inner, int64_t repeat) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total = outer * dim_size * inner;
    if (idx >= total) {
        return;
    }

    int64_t i = idx / inner;
    int64_t j = idx % inner;

    int64_t o = i / dim_size;
    int64_t di = i % dim_size;

    float sum = 0.0f;
    for (int64_t r = 0; r < repeat; ++r) {
        int64_t out_idx = ((o * dim_size * repeat + di * repeat + r) * inner) + j;
        sum += grad_output[out_idx];
    }
    grad_input[idx] = sum;
}

std::shared_ptr<Tensor> RepeatInterleaveBackward(const std::shared_ptr<Tensor> &grad_output,
                                                 const std::vector<int64_t> &input_dims, int64_t dim) {
    CHECK_GE(dim, 0);
    CHECK_LT(dim, input_dims.size());

    const int64_t outer = std::accumulate(input_dims.begin(), input_dims.begin() + dim, 1, std::multiplies<int64_t>());
    const int64_t inner
        = std::accumulate(input_dims.begin() + dim + 1, input_dims.end(), 1, std::multiplies<int64_t>());
    const int64_t dim_size = input_dims[dim];

    int64_t repeat = grad_output->Dims()[dim] / dim_size;
    CHECK_EQ(grad_output->Dims()[dim], dim_size * repeat);

    auto grad_input = std::make_shared<Tensor>(input_dims, grad_output->Dtype(), grad_output->GetDevice());
    grad_input->Fill<float>(0.0f);

    const float *grad_out_ptr = static_cast<const float *>(grad_output->DataPtr());
    float *grad_in_ptr = static_cast<float *>(grad_input->DataPtr());

    int64_t total_elements = outer * dim_size * inner;
    int threads_per_block = 256;
    int num_blocks = (total_elements + threads_per_block - 1) / threads_per_block;
    RepeatInterleaveBackwardKernel<<<num_blocks, threads_per_block>>>(grad_out_ptr, grad_in_ptr, outer, dim_size, inner,
                                                                      repeat);

    return grad_input;
}
} // namespace infini_train::kernels::cuda

#define REGISTER_CUDA_TRANSFORM_KERNEL(kernel_name)                                                                    \
    REGISTER_KERNEL(infini_train::DeviceType::kCUDA, kernel_name, infini_train::kernels::cuda::kernel_name)

REGISTER_CUDA_TRANSFORM_KERNEL(TrilForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TrilBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(TriuForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TriuBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(TransposeForward)
REGISTER_CUDA_TRANSFORM_KERNEL(TransposeBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(MaskForward)
REGISTER_CUDA_TRANSFORM_KERNEL(MaskBackward)
REGISTER_CUDA_TRANSFORM_KERNEL(RepeatInterleaveForward)
REGISTER_CUDA_TRANSFORM_KERNEL(RepeatInterleaveBackward)

#undef REGISTER_CUDA_TRANSFORM_KERNEL
